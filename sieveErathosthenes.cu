
#include <hip/hip_runtime.h>
#include <stdio.h>

__inline__ __device__ unsigned long long __warpReduce(unsigned long long sum){
	sum += __shfl_xor_sync(0xffffffff, sum, 1);
	sum += __shfl_xor_sync(0xffffffff, sum, 2);
	sum += __shfl_xor_sync(0xffffffff, sum, 4);
	sum += __shfl_xor_sync(0xffffffff, sum, 8);
	sum += __shfl_xor_sync(0xffffffff, sum, 16);
	return sum;
}

__global__ void __sievePrime(bool* tempStore, const unsigned long total){
	unsigned long idx = (threadIdx.x + blockIdx.x * blockDim.x << 1) + 3;
	for(unsigned long i = idx * idx; i <= (total + 1) << 1; i += idx << 4){
		tempStore[(i - 3) >> 1] = false;
		tempStore[((idx << 1) + i - 3) >> 1] = false;
		tempStore[((idx << 2) + i - 3) >> 1] = false;
		tempStore[(6 * idx + i - 3) >> 1] = false;
		tempStore[((idx << 3) + i - 3) >> 1] = false;
		tempStore[(10 * idx + i - 3) >> 1] = false;
		tempStore[(12 * idx + i - 3) >> 1] = false;
		tempStore[(14 * idx + i - 3) >> 1] = false;
	}
}

__global__ void __parallelSum(bool* tempStore, const unsigned long total, unsigned long long* primeSum){
	unsigned long idx = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ unsigned char warpSum[32];
	unsigned char warpID = threadIdx.x / warpSize, laneID = threadIdx.x % warpSize; 
	unsigned long long sum = (idx < total)? tempStore[idx] : 0;
	sum = __warpReduce(sum);
	if (laneID == 0) warpSum[warpID] = sum;
       __syncthreads();
       sum = (threadIdx.x < warpSize)? warpSum[laneID] : 0;
       if (warpID == 0) sum = __warpReduce(sum);
       if (threadIdx.x == 0) atomicAdd(primeSum, sum);
}

unsigned long long prime(unsigned long long total){
	bool *tempStore;
	total = total / 2 - 1;
	unsigned long long *devPrimeSum, *hostPrimeSum;
	unsigned long launch = sqrt(total);
	hostPrimeSum = (unsigned long long*)malloc(sizeof(unsigned long long));
	hipMalloc(&devPrimeSum, sizeof(unsigned long long));
	hipMalloc(&tempStore, total * sizeof(bool));
	hipMemset(tempStore, true, total * sizeof(bool));
	__sievePrime <<< launch, 1 >>> (tempStore, total);
	hipDeviceSynchronize();
	__parallelSum <<< total / 128 + 1, 128 >>> (tempStore, total, devPrimeSum);
	hipMemcpy(hostPrimeSum, devPrimeSum, sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipFree(tempStore);
	hipFree(devPrimeSum);
	return *hostPrimeSum + 1;
}

int main(){
	printf("Total primes for 1 billion numbers -> %d\n", prime(1000000000));
	return 0;
}
